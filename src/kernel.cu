#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstdio>
#include <ctime>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>
#include "vector.h"

const half ACOalpha = 0.2;//increase
const half ACOdelta = 0.1;//decrease
const half ACOgamma = 0.1;//minval
const int block_size = 10;
const int thread_size = 32;

__device__ half& tmat(half *matrix, size_t x, size_t y){
	return x>y ? matrix[y*N+x] : matrix[x*N+y];
}
__device__ char& tmat(char *matrix, size_t x, size_t y){
	return x>y ? matrix[y*N+x] : matrix[x*N+y];
}

__global__ void clique_kernel(size_t *A, size_t N, char *device_graph, half *device_pheromone, hiprandState *states, unsigned int seed) {
	size_t id = blockIdx.x*blockDim.x+threadIdx.x;
	actual_device_vector<size_t > B, C;
    hiprand_init(seed*id, id, 0, &states[id]);
	size_t  startIdx = (1-hiprand_uniform(&states[id]))*N;
	size_t current = startIdx;
	for(size_t i = 0; i < N; i++) if(i!=startIdx && tmat(device_graph, startIdx, i)) B.push_back(i);
	C.push_back(startIdx); //END SETUP
	while(B.size()>0){ //MAIN LOOP
		half norm = 0;
		for(size_t i = 0; i < B.size(); norm += tmat(device_pheromone, current, B[i++]));
		size_t chosen = 0;
		float radom = (1-hiprand_uniform(&states[id]));
		while(chosen < B.size() && radom<=0) radom-= tmat(device_pheromone, current, B[chosen++], N)/norm; //NEXT VERTEX PICKED
		for(size_t i = 0; i < B.size(); i++) if(i==chosen || !tmat(device_graph, chosen, B[i])) B.erase(i); //REMOVE NON-NEIGHBORING
		current = chosen;
		C.push_back(chosen);
	}
	for(size_t i = 0; i < C.size()-1; i++) tmat(device_pheromone, C[i], C[i+1])=tmat(device_pheromone, C[i], C[i+1])+(half)ACOalpha;
	A[id]=C.size();
}
__global__ void evaporation_kernel(size_t N, half *device_pheromone){
	size_t id = blockIdx.x *blockDim.x + threadIdx.x;
	if(id<N*N) device_pheromone[id] = device_pheromone[id]<= ACOgamma+ACOdelta ? (half)ACOgamma : device_pheromone[id]-(half)ACOdelta;
}
extern "C" int anthill(char **graph, size_t N, size_t M){
	hiprandState *states;
	char *device_graph;
	half *device_pheromone;
	size_t *results, *host_results=new size_t[block_size*thread_size], max = 0;
	hipMalloc(&states, block_size*thread_size*sizeof(hiprandState));
	
	hipMalloc(&device_graph, N*N*sizeof(char));
	hipMalloc(&device_pheromone, N*N*sizeof(unsigned short));
	hipMemcpy(device_graph, graph[0], N*N*sizeof(char), hipMemcpyHostToDevice); //graph initialized
	
	hipMalloc(&results, block_size*thread_size*sizeof(size_t));
	for(size_t i = 0; i < M; i++){
		evaporation_kernel<<<N*N/thread_size,thread_size>>>(N, device_pheromone);
		clique_kernel<<<block_size,thread_size>>>(results, N, device_graph, device_pheromone, states, (unsigned int)time(NULL));
		hipMemcpy(host_results, results, block_size*thread_size*sizeof(size_t), hipMemcpyDeviceToHost);
		for(size_t j = 0; j < block_size; j++) if(max<host_results[j]) max = host_results[j];
	}
	delete[] host_results;
	hipFree(results);
	
	hipFree(device_pheromone);
	hipFree(device_graph);
	hipFree(states);
	return max;
}