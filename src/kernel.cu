#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>
#include "vector.h"

const float ACOalpha = 0.2;//increase
const float ACOdelta = 0.1;//decrease
const float ACOgamma = 0.1;//minval
const int block_size = 10;

__global__ void clique_kernel(int *A, int N, char **device_graph, float **device_pheromone, hiprandState *states) {
	int id = blockIdx.x*blockDim.x;
    hiprand_init(1342, id, 0, &states[id]);
	int startIdx = hiprand_uniform(&states[id])*N;
	actual_device_vector<int> C(N/2), B(N/2);
	for(int i = 0; i < N; i++) if(device_graph[startIdx][i]) B.push_back(i);
	C.push_back(startIdx); //END SETUP
	int current = startIdx;
	while(B.size()>0){ //MAIN LOOP
		float norm = 0.0f;
		for(int i = 0; i < B.size(); i++) norm += device_pheromone[current][B[i]];
		int chosen = 0;
		float radom = hiprand_uniform(&states[id]);
		for(chosen=0;chosen < B.size() || radom<=0;chosen++,radom-= device_pheromone[current][B[chosen]]/norm); //NEXT VERTEX PICKED
		for(int i = 0; i < B.size(); i++) if(!device_graph[chosen][B[i]]) B.erase(i); //REMOVE NON-NEIGHBORING
		current = chosen;
		C.push_back(chosen);
	}
	for(int i = 0; i < C.size()-1; i++) {
		device_pheromone[C[i]][C[i+1]]=device_pheromone[C[i]][C[i+1]]+ACOalpha;
		device_pheromone[C[i+1]][C[i]]=device_pheromone[C[i]][C[i+1]];
	}
	A[id]=C.size();
}
__global__ void evaporation_kernel(int N, float **device_pheromone){
	int row = blockIdx.x *blockDim.x + threadIdx.x;
	for(int i = 0; i<N; i++) {
		device_pheromone[row][i] -= ACOdelta;
		if(device_pheromone[row][i]<ACOgamma) device_pheromone[row][i]=ACOgamma;
	}
}

extern "C" int anthill(char **graph, int N, int M){
	hiprandState *states;
	char **device_graph;
	float **device_pheromone;
	hipMalloc(&states, N*sizeof(hiprandState));
	hipMalloc(&device_graph, N*N*sizeof(char));
	hipMalloc(&device_pheromone, N*N*sizeof(float));
	hipMemcpy(device_graph, graph, N*N*sizeof(char), hipMemcpyHostToDevice); //graph initialized
	int *results, *host_results=new int[block_size], max = 0;
	hipMalloc(&results, block_size*sizeof(int));
	for(int i = 0; i < M; i++){
		evaporation_kernel<<<N,1>>>(N, device_pheromone);
		clique_kernel<<<block_size,1>>>(results, N, device_graph, device_pheromone, states);
		printf("%s ", hipGetErrorName(hipMemcpy(host_results, results, block_size*sizeof(int), hipMemcpyDeviceToHost)));
		for(int i = 0; i < block_size; i++) if(max<host_results[i]) max = host_results[i];
	}
	delete[] host_results;
	hipFree(results);
	hipFree(device_pheromone);
	hipFree(device_graph);
	hipFree(states);
	return max;
}