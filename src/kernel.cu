#include "hip/hip_runtime.h"
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/fill.h> 

const float ACOalpha = 0.2;//increase
const float ACOdelta = 0.1;//decrease
const float ACOgamma = 0.1;//minval
const int block_size = 10;
const int iteration_count = 10000;

__global__ void clique_kernel(int *A, int N, char **device_graph, float **device_pheromone, hiprandState *state) {
	int startIdx = hiprand_uniform(&state)*N;
	thrust::device_vector<int> C(N/2), B(N/2);
	for(int i = 0; i < N; i++) if(graph[startIdx][i]) B.push_back(i);
	C.push_back(startIdx); //END SETUP
	int current = startIdx;
	while(B.size()>0){ //MAIN LOOP
		float norm = 0.0f;
		for(int i = 0; i < B.size(); i++) norm += device_pheromone[current][B[i]];
		int chosen = 0;
		float radom = hiprand_uniform(&state);
		for(chosen=0;chosen < B.size() || radom<=0;chosen++,radom-= device_pheromone[current][B[chosen]]/norm); //NEXT VERTEX PICKED
		for(int i = 0; i < B.size(); i++) if(!graph[chosen][B[i]]) B.erase(B.begin()+i); //REMOVE NON-NEIGHBORING
		current = chosen;
		C.push_back(chosen);
	}
	for(int i = 0; i < C.size()-1; i++) {
		device_pheromone[C[i]][C[i+1]]=device_pheromone[C[i]][C[i+1]]+ACOalpha;
		device_pheromone[C[i+1]][C[i]]=device_pheromone[C[i]][C[i+1]];
	}
	A[blockIdx.x*blockDim.x]=C.size();
}
__global__ void evaporation_kernel(int N, float **device_pheromone){
	int row = blockIdx.x *blockDim.x + threadIdx.x;
	for(int i = 0; i<N; i++) {
		device_pheromone[row][i] -= ACOdelta;
		if(device_pheromone[row][i]<ACOgamma) device_pheromone[row][i]=ACOgamma;
	}
}
extern "C" int anthill(char **graph, int N, int M){
	hiprandState *state;
	char **device_graph;
	float **device_pheromone;
	hipMalloc(&state, sizeof(hiprandState));
	hipMalloc(&device_graph, N*sizeof(char*));
	hipMalloc(&device_pheromone, N*sizeof(float*));
    hiprand_init(time(NULL), i, 0, state);
	void **temp = malloc(N*sizeof(char*)), **temp2 = malloc(N*sizeof(float*));
	for(int i = 0; i < N; i++) {
		hipMalloc(&temp[i], N*sizeof(char));
		hipMemcpy(temp[i], graph[i], N, hipMemcpyHostToDevice);
		hipMalloc(&temp2[i], N*sizeof(float));
		hipMemset(temp2[i], ACOgamma, N);
	}
	hipMemcpy(device_graph, temp, N, hipMemcpyHostToDevice); //graph initialized
	hipMemcpy(device_pheromone, temp2, N, hipMemcpyHostToDevice); //device_pheromone initialized
	int *results, *host_results=malloc(block_size*sizeof(int)), max = 0;
	hipMalloc(&results, block_size*sizeof(int));
	for(int i = 0; i < M; i++){
		clique_kernel<<<block_size,1>>>(results, N, device_graph, device_pheromone, state);
		evaporation_kernel<<<N,1>>>(N, device_pheromone);
		hipMemcpy(host_results, results, N, hipMemcpyDeviceToHost);
		for(int i = 0; i < block_size; i++) if(max<host_results[i]) max = host_results[i];
	}
	hipFree(results);
	for(int i = 0; i < N; i++){
		hipFree(temp[i]);
		hipFree(temp2[i]);
	}
	hipFree(device_pheromone);
	hipFree(device_graph);
	hipFree(state);
	return max;
}