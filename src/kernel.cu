#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstdio>
#include <ctime>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>
#include "vector.h"

const char ACOalpha = 2;//increase
const char ACOdelta = 1;//decrease
const char ACOgamma = 1;//minval
const int block_size = 10;
const int thread_size = 10;

__global__ void clique_kernel(int *A, int N, char *device_graph, char *device_pheromone, hiprandState *states, unsigned int seed) {
	int id = blockIdx.x*blockDim.x;
    hiprand_init(seed*id, id, 0, &states[id]);
	int startIdx = (1-hiprand_uniform(&states[id]))*N;
	actual_device_vector<int> C(N/2), B(N/2);
	for(int i = 0; i < N; i++) if(i!=startIdx && device_graph[startIdx*N+i]) B.push_back(i);
	C.push_back(startIdx); //END SETUP
	int current = startIdx;
	while(B.size()>0){ //MAIN LOOP
		float norm = 0.0f;
		for(int i = 0; i < B.size(); i++) norm += device_pheromone[current*N+B[i]];
		int chosen = 0;
		float radom = (1-hiprand_uniform(&states[id]));
		for(chosen=0;chosen < B.size() && radom<=0;chosen++,radom-= device_pheromone[current*N+B[chosen]]/norm); //NEXT VERTEX PICKED
		for(int i = 0; i < B.size(); i++) if(i==chosen || !device_graph[chosen*N+B[i]]) B.erase(i); //REMOVE NON-NEIGHBORING
		current = chosen;
		C.push_back(chosen);
	}
	for(int i = 0; i < C.size()-1; i++) {
		device_pheromone[C[i]*N+C[i+1]]=device_pheromone[C[i]*N+C[i+1]]+ACOalpha;
		device_pheromone[C[i+1]*N+C[i]]=device_pheromone[C[i]*N+C[i+1]];
	}
	A[id]=C.size();
}
__global__ void evaporation_kernel(int N, char *device_pheromone){
	int row = blockIdx.x *blockDim.x + threadIdx.x*thread_size;
	for(int i = 0; i<N/thread_size; i++) {
		device_pheromone[row*N+i] -= ACOdelta;
		if(device_pheromone[row*N+i]<ACOgamma) device_pheromone[row*N+i]=ACOgamma;
	}
}
extern "C" int anthill(char **graph, int N, int M){
	hiprandState *states;
	char *device_graph;
	char *device_pheromone;
	hipMalloc(&states, N*sizeof(hiprandState));
	hipMalloc(&device_graph, N*N*sizeof(char));
	hipMalloc(&device_pheromone, N*N*sizeof(char));
	hipMemcpy(device_graph, graph[0], N*N*sizeof(char), hipMemcpyHostToDevice); //graph initialized
	int *results, *host_results=new int[block_size*thread_size], max = 0;
	hipMalloc(&results, block_size*thread_size*sizeof(int));
	for(int i = 0; i < M; i++){
		evaporation_kernel<<<N,thread_size>>>(N, device_pheromone);
		clique_kernel<<<block_size,thread_size>>>(results, N, device_graph, device_pheromone, states, (unsigned int)time(NULL));
		hipMemcpy(host_results, results, block_size*thread_size*sizeof(int), hipMemcpyDeviceToHost);
		for(int i = 0; i < block_size; i++) if(max<host_results[i]) max = host_results[i];
	}
	delete[] host_results;
	hipFree(results);
	hipFree(device_pheromone);
	hipFree(device_graph);
	hipFree(states);
	return max;
}